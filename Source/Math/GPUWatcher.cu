//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUWatcher.h"
#ifdef __HIP_PLATFORM_NVCC__
    #include <hip/hip_runtime.h>
#endif
#include <hip/hip_runtime.h>

int GPUWatcher::GetGPUIdWithTheMostFreeMemory()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess || deviceCount == 0)
    {
        return -1;
    }
    int curDev = 0;
    size_t curMemory = 0;
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        size_t freeMem = GetFreeMemoryOnCUDADevice(dev);
        if (freeMem > curMemory)
        {
            curMemory = freeMem;
            curDev = dev;
        }
    }
    return curDev;
}

size_t GPUWatcher::GetFreeMemoryOnCUDADevice(int devId)
{
    hipError_t result = hipSetDevice(devId);
    if (result != hipSuccess)
    {
        return 0;
    }
    // get the amount of free memory on the graphics card
    size_t free = 0;
    size_t total = 0;
    result = hipMemGetInfo(&free, &total);
    if (result != hipSuccess)
    {
        return 0;
    }
    else
        return free;
}

GPUWatcher::GPUWatcher(void)
{
}

GPUWatcher::~GPUWatcher(void)
{
}

#endif // CPUONLY
